#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;

#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
    CUDA_CALL(hipEventCreate(&cuda_timer_start));
    CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
    CUDA_CALL(hipEventDestroy(cuda_timer_start));
    CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
    hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
    TIMER_T ms;
    hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
    hipEventSynchronize(cuda_timer_stop);

    hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
    return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

#define BLOCK_SIZE 64
#define DATA_SIZE (1 << 26)

TIMER_T compute_time = 0;
TIMER_T device_time = 0;
#define N_EQUATIONS 1 << 26
int N;
float* A;
float* B;
float* C;
float* X0;
float* X1;
float* FX0;
float* FX1;
float* X0_gpu;
float* X1_gpu;
float* FX0_gpu;
float* FX1_gpu;

hipError_t Equation_GPU(float* A, float* B, float* C, float* X0, float* X1, float* fX0, float* fX1, int n);

__global__ void Equation_Kernel(float* A, float* B, float* C, float* X0, float* X1, float* FX0, float* FX1)
{
    float a, b, c, d, x0, x1, tmp;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = gridDim.x * blockDim.x * row + col;
        a = A[tid]; b = B[tid]; c = C[tid];
        d = sqrtf(b * b - 4.0f * a * c);
        tmp = 1.0f / (2.0f * a);
        X0[tid] = x0 = (-b - d) * tmp;
        X1[tid] = x1 = (-b + d) * tmp;
        FX0[tid] = (a * x0 + b) * x0 + c;
        FX1[tid] = (a * x1 + b) * x1 + c;
}


void Equation_CPU(float *A, float * B, float *C, float *X0, float *X1, float *FX0, float *FX1, int n) {
    int i;
    float a, b, c, d, x0, x1, tmp;
    for (i = 0; i < n; i++) {
        a = A[i]; b = B[i]; c = C[i];
        d = sqrtf(b * b - 4.0f * a * c);
        tmp = 1.0f / (2.0f * a);
        X0[i] = x0 = (-b - d) * tmp;
        X1[i] = x1 = (-b + d) * tmp;
        FX0[i] = (a * x0 + b) * x0 + c;
        FX1[i] = (a * x1 + b) * x1 + c;
    }
}

void init_bin_file() {
    srand((unsigned)time(NULL));
    N = N_EQUATIONS;
    A = (float*)malloc(sizeof(float) * N);
    B = (float*)malloc(sizeof(float) * N);
    C = (float*)malloc(sizeof(float) * N);
    X0 = (float*)malloc(sizeof(float) * N);
    X1 = (float*)malloc(sizeof(float) * N);
    FX0 = (float*)malloc(sizeof(float) * N);
    FX1 = (float*)malloc(sizeof(float) * N);
    X0_gpu = (float*)malloc(sizeof(float) * N);
    X1_gpu = (float*)malloc(sizeof(float) * N);
    FX0_gpu = (float*)malloc(sizeof(float) * N);
    FX1_gpu = (float*)malloc(sizeof(float) * N);//���� �Ҵ�.
    printf("***Binary File init Start!!\n");
    FILE *fp1 = fopen("A.bin", "wb");
    FILE* fp2 = fopen("B.bin", "wb");
    FILE* fp3 = fopen("C.bin", "wb");


    float random_a;
    float random_b;
    float random_c;
    for (int i = 0; i < N; i++) {
        random_a = (float)(((rand() %10)));
        random_c = (float)(((rand() %10)*(-1)));
        random_b = (float)(((rand() %10)));
        fwrite(&random_a, sizeof(float), 1, fp1);
        fwrite(&random_b, sizeof(float), 1, fp2);
        fwrite(&random_c, sizeof(float), 1, fp3);
    }

    fclose(fp1);
    fclose(fp2);
    fclose(fp3);
    printf("***Binary File init End!!\n\n");
}

void read_bin_file()
{
    printf("***Binary File Read Start!!\n");
    FILE *fp1 = fopen("a.bin", "rb");
    FILE* fp2 = fopen("b.bin", "rb");
    FILE* fp3 = fopen("c.bin", "rb");
    /*Todo*/

    int i;
    for (i = 0; i < N; i++)//N����ŭ ���� �о�� h_Fibonacci_number�� ����
    {
        fread(&A[i], sizeof(float), 1, fp1);
        fread(&B[i], sizeof(float), 1, fp2);
        fread(&C[i], sizeof(float), 1, fp3);
    }


    fclose(fp1);
    fclose(fp2);
    fclose(fp3);
    printf("***Binary File Read End!!\n\n");
}

int main()
{
    init_bin_file();
    read_bin_file();

    printf("The problem size is %d.\n", N);
    int i;

    //CPU 
    printf("***Equation_CPU Start!!\n");
    CHECK_TIME_START;
    Equation_CPU(A, B, C, X0, X1, FX0, FX1, N);
    CHECK_TIME_END(compute_time);
    printf("***Equation_CPU End!!\n");

    printf("CPU time = %.6f\n\n", compute_time);
    //GPU

    printf("***Equation_GPU Start!!\n");
    Equation_GPU(A, B, C, X0_gpu, X1_gpu, FX0_gpu, FX1_gpu, N);
    printf("***Equation_GPU End!!\n");

    printf("GPU time = %.6f\n", device_time);

    for (i = 0; i < N; i++)
    {
        if (fabs(X0[i] - X0_gpu[i])>0.0001 || fabs(X1[i] - X1_gpu[i])>0.0001)
        {
            printf("x0[i] %f, x0_gpu[i] %f\n", X0[i], X0_gpu[i]);
            printf("x1[i] %f, x1_gpu[i] %f\n", X1[i], X1_gpu[i]);
            break;
        }
        if (fabs(FX0_gpu[i]) > 0.0001 || fabs(FX1_gpu[i]) > 0.0001)
        {
            printf("fx0_gpu[i] %f\n", FX0_gpu[i]);
            printf("fx1_gpu[i] %f\n", FX1_gpu[i]);
            break;
        }
    }

    if (i == N)
        printf("***Kernel execution Success!!\n\n");

    // Write the output array into the output file.
    FILE *fp1 = fopen("X0.bin", "wb");
    if (!fp1) {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }
    FILE* fp2 = fopen("X1.bin", "wb");
    if (!fp2) {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }
    FILE* fp3 = fopen("FX0.bin", "wb");
    if (!fp3) {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }
    FILE* fp4 = fopen("FX1.bin", "wb");
    if (!fp4) {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }

    fwrite(&X0_gpu, sizeof(float), N, fp1);
    fwrite(&X1_gpu, sizeof(float), N, fp2);
    fwrite(&FX0_gpu, sizeof(float), N, fp3);
    fwrite(&FX1_gpu, sizeof(float), N, fp4);

    fclose(fp1);
    fclose(fp2);
    fclose(fp3);
    fclose(fp4);
    printf("end!!\n\n");

    return 0;
}


hipError_t Equation_GPU(float* A, float* B, float* C, float* X0_gpu, float* X1_gpu, float* FX0_gpu, float* FX1_gpu, int n)
{
    CHECK_TIME_INIT_GPU();

    hipError_t cudaStatus;
    /*Todo*/
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }/////////////  if(cu.....  ==CUDA_CALL
    float* d_a;
    float* d_b;
    float* d_c;
    float* d_x0;
    float* d_x1;
    float* d_fx0;
    float* d_fx1;
    size_t size=N*sizeof(float);

    CUDA_CALL(hipMalloc(&d_a, size))//gpu�� a�� �����ϱ� ���Ͽ� �޸� �Ҵ�.
        CUDA_CALL(hipMemcpy(d_a, A, size, hipMemcpyHostToDevice))//gpu(d_a)�� a copy��.
        CUDA_CALL(hipMalloc(&d_b, size))//gpu�� b�� �����ϱ� ���Ͽ� �޸� �Ҵ�.
        CUDA_CALL(hipMemcpy(d_b, B, size, hipMemcpyHostToDevice))//gpu(d_b)�� b copy��.
        CUDA_CALL(hipMalloc(&d_c, size))//gpu�� c�� �����ϱ� ���Ͽ� �޸� �Ҵ�.
        CUDA_CALL(hipMemcpy(d_c, C, size, hipMemcpyHostToDevice))//gpu(d_c)�� c copy��.
        CUDA_CALL(hipMalloc(&d_x0, size))// gpu���� ������� ����� memory �Ҵ�.
        CUDA_CALL(hipMalloc(&d_x1, size))// gpu���� ������� ����� memory �Ҵ�.
        CUDA_CALL(hipMalloc(&d_fx0, size))// gpu���� ������� ����� memory �Ҵ�.
        CUDA_CALL(hipMalloc(&d_fx1, size))// gpu���� ������� ����� memory �Ҵ�.



        // Assume that width and height are multiples of BLOCK SIZE.
        dim3 dimBlock(BLOCK_SIZE);//block dimension 1����, block size
        dim3 dimGrid(N/BLOCK_SIZE);//n/block_size�� grid�� dimension
    CHECK_TIME_START_GPU()
        Equation_Kernel << < dimGrid, dimBlock >> > (d_a, d_b, d_c, d_x0, d_x1, d_fx0, d_fx1);//kernel ����
    CHECK_TIME_END_GPU(device_time)
        CUDA_CALL(hipGetLastError())

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        CUDA_CALL(hipDeviceSynchronize())
        CUDA_CALL(hipMemcpy(X0_gpu, d_x0, size, hipMemcpyDeviceToHost))//gpu�޸𸮷κ��� ����� ī��
        CUDA_CALL(hipMemcpy(X1_gpu, d_x1, size, hipMemcpyDeviceToHost))//gpu�޸𸮷κ��� ����� ī��
        CUDA_CALL(hipMemcpy(FX0_gpu, d_fx0, size, hipMemcpyDeviceToHost))//gpu�޸𸮷κ��� ����� ī��
        CUDA_CALL(hipMemcpy(FX1_gpu, d_fx1, size, hipMemcpyDeviceToHost))//gpu�޸𸮷κ��� ����� ī��

        Error:
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_x0);
    hipFree(d_x1);
    hipFree(d_fx0);
    hipFree(d_fx1);
    CHECK_TIME_DEST_GPU();
    return cudaStatus;
}