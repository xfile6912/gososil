#include "hip/hip_runtime.h"
#pragma once

#include "cuda_code.cuh"

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL(hipEventDestroy(cuda_timer_start));
	CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

#define Window 2
#define BLOCK_SIZE 512												// CUDA Ŀ�� thread block ������

#define BLOCK_WIDTH (1 << 3)
#define BLOCK_HEIGHT (BLOCK_SIZE / BLOCK_WIDTH)
__constant__ float constant_gaussian_kernel[ 25 ];

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Gaussian ���͸��� �ϴ� Ŀ��
//	shared memory�� ������� �ʴ´�
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void Gaussian_kernel_no_shared(IN unsigned char *d_bitmaps, OUT unsigned char *d_Gaussian, long width, long height) {
	/*Todo*/
	const unsigned block_id = blockIdx.y * gridDim.x + blockIdx.x;
	const unsigned thread_id = threadIdx.y * blockDim.x + threadIdx.x;
	const unsigned id = block_id * BLOCK_SIZE + thread_id;
	int w = 2;
	int i, j;
	int row = id / width;
	int col = id % width;
	d_Gaussian[id] = 0;
	for (i = -w; i <= w; i++)
	{
		for (j = -w; j <= w; j++)
		{
			if (row + i >= 0 && col + j >= 0 && row + i < height && col + j < width)//�����ȿ� ���� ��
			{
				d_Gaussian[id] += constant_gaussian_kernel[(i + w) * 5 + j + w] * d_bitmaps[(row+i)*width+(col+j)];//i+w�� 0<i+w<w0, j+w�� 0<j+w<w0
			}
			else//���� �ۿ� ������
			{
				d_Gaussian[id] += 0;
			}
		}
	}


}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Gaussian ���͸��� �ϴ� Ŀ��
//	shared memory�� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
extern __shared__ unsigned char sharedBuffer[];
__global__ void Gaussian_kernel_shared(INOUT unsigned char *d_bitmaps, OUT unsigned char *d_Gaussian, long width, long height) {
	/*Todo*/
	const unsigned block_id = blockIdx.y * gridDim.x + blockIdx.x;
	const unsigned thread_id = threadIdx.y * blockDim.x + threadIdx.x;
	const unsigned id = block_id * BLOCK_SIZE + thread_id;
	int row=id/width;
	int col=id%width;
	int sharedRowSize = BLOCK_SIZE + 4;//5���� row�� ����µ� row size�� 2+ BLOCK_SIZE+2 ��ŭ.
	int i;
	if (thread_id == 0)
	{
		for (i = 0; i < 2; i++)
		{
			if (col + i < 2)
			{
					sharedBuffer[i + sharedRowSize * 0] = 0;
					sharedBuffer[i + sharedRowSize * 1] = 0;
					sharedBuffer[i + sharedRowSize * 2] = 0;//id �ڱ� �ڽ�
					sharedBuffer[i + sharedRowSize * 3] = 0;
					sharedBuffer[i + sharedRowSize * 4] = 0;
			}
			else
			{
				if (row - 2 >= 0)//id���� ��ĭ ���� row
					sharedBuffer[i + sharedRowSize * 0] = d_bitmaps[(row - 2) * width + col+i-2];
				else
					sharedBuffer[i + sharedRowSize * 0] = 0;

				if (row - 1 >= 0)//id���� ��ĭ ���� row
					sharedBuffer[i + sharedRowSize * 1] = d_bitmaps[(row - 1) * width + col + i - 2];
				else
					sharedBuffer[i + sharedRowSize * 1] = 0;

				sharedBuffer[i + sharedRowSize * 2] = d_bitmaps[id + i - 2];//id �ڱ� �ڽ�

				if (row + 1 < height)
					sharedBuffer[i + sharedRowSize * 3] = d_bitmaps[(row + 1) * width + col + i - 2];//id���� 1ĭ �Ʒ��� row
				else
					sharedBuffer[i+ sharedRowSize * 3] = 0;

				if (row + 2 < height)
					sharedBuffer[i + sharedRowSize * 4] = d_bitmaps[(row + 2) * width + col + i - 2];//id���� ��ĭ �Ʒ��� row
				else
					sharedBuffer[i + sharedRowSize * 4] = 0;
			}
		}
	}
	if (thread_id == BLOCK_SIZE - 1)
	{
		for (i = 1; i <= 2; i++)
		{
			if (col + i >= width)
			{
				sharedBuffer[thread_id + 2+i + sharedRowSize * 0] = 0;
				sharedBuffer[thread_id + 2+i + sharedRowSize * 1] = 0;
				sharedBuffer[thread_id + 2+i + sharedRowSize * 2] = 0;//id �ڱ� �ڽ�
				sharedBuffer[thread_id + 2+i + sharedRowSize * 3] = 0;
				sharedBuffer[thread_id + 2+i + sharedRowSize * 4] = 0;
			}
			else
			{
				if (row - 2 >= 0)//id���� ��ĭ ���� row
					sharedBuffer[thread_id + 2+i + sharedRowSize * 0] = d_bitmaps[(row - 2) * width + col + i];
				else
					sharedBuffer[thread_id + 2+i + sharedRowSize * 0] = 0;

				if (row - 1 >= 0)//id���� ��ĭ ���� row
					sharedBuffer[thread_id + 2+i + sharedRowSize * 1] = d_bitmaps[(row - 1) * width + col + i];
				else
					sharedBuffer[thread_id + 2+i + sharedRowSize * 1] = 0;

				sharedBuffer[thread_id + 2+i + sharedRowSize * 2] = d_bitmaps[id + i];//id �ڱ� �ڽ�

				if (row + 1 < height)
					sharedBuffer[thread_id + 2+i + sharedRowSize * 3] = d_bitmaps[(row + 1) * width + col + i];//id���� 1ĭ �Ʒ��� row
				else
					sharedBuffer[thread_id + 2+i + sharedRowSize * 3] = 0;

				if (row + 2 < height)
					sharedBuffer[thread_id + 2+i + sharedRowSize * 4] = d_bitmaps[(row + 2) * width + col + i];//id���� ��ĭ �Ʒ��� row
				else
					sharedBuffer[thread_id + 2+i + sharedRowSize * 4] = 0;
			}
		}
	}

	if (row - 2 >= 0)//id���� ��ĭ ���� row
		sharedBuffer[thread_id + 2 + sharedRowSize * 0] = d_bitmaps[(row - 2) * width + col];
	else
		sharedBuffer[thread_id + 2 + sharedRowSize * 0] = 0;

	if (row - 1 >= 0)//id���� ��ĭ ���� row
		sharedBuffer[thread_id + 2 + sharedRowSize * 1] = d_bitmaps[(row - 1) * width + col];
	else
		sharedBuffer[thread_id + 2 + sharedRowSize * 1] = 0;
	sharedBuffer[(thread_id + 2) + sharedRowSize * 2] = d_bitmaps[id];//id �ڱ� �ڽ�
	
	if (row + 1 < height)
		sharedBuffer[thread_id + 2 + sharedRowSize * 3] = d_bitmaps[(row + 1) * width + col];//id���� 1ĭ �Ʒ��� row
	else
		sharedBuffer[thread_id + 2 + sharedRowSize * 3] = 0;
	if (row +2 < height)
		sharedBuffer[thread_id + 2 + sharedRowSize * 4] = d_bitmaps[(row + 2) * width + col];//id���� ��ĭ �Ʒ��� row
	else
		sharedBuffer[thread_id + 2 + sharedRowSize * 4] = 0;

	__syncthreads();
	
	int j;
	d_Gaussian[id] = 0;
	for (i = 0; i < 5; i++)
	{
		for (j = 0; j < 5; j++)
		{
			d_Gaussian[id] += constant_gaussian_kernel[i * 5 + j] * sharedBuffer[i * sharedRowSize + thread_id + j];
		}
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Constant variable �� gaussian kernel�� �����ϴ� �Լ�
//	�Ŀ� gaussian filtering ���� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void Set_Gaussain_Kernel(){
	float _1 = 1.0f / 256.0f;
	float _4 = _1 * 4;
	float _6 = _1 * 6;
	float _16 = _1 * 16;
	float _24 = _1 * 24;
	float _36 = _1 * 36;

	float *p_gaussian_kernel = new float[25];

	p_gaussian_kernel[0] = p_gaussian_kernel[4] = p_gaussian_kernel[20] = p_gaussian_kernel[24] = _1;
	p_gaussian_kernel[1] = p_gaussian_kernel[3] = p_gaussian_kernel[5] = p_gaussian_kernel[9]= _4;
	p_gaussian_kernel[15] = p_gaussian_kernel[19] = p_gaussian_kernel[21] = p_gaussian_kernel[23] = _4;
	p_gaussian_kernel[2] = p_gaussian_kernel[10] = p_gaussian_kernel[14] = p_gaussian_kernel[22] = _6;
	p_gaussian_kernel[6] = p_gaussian_kernel[8] = p_gaussian_kernel[16] = p_gaussian_kernel[18] = _16;
	p_gaussian_kernel[7] = p_gaussian_kernel[11] =p_gaussian_kernel[13] = p_gaussian_kernel[17] = _24;
	p_gaussian_kernel[12] = _36;

	hipMemcpyToSymbol(HIP_SYMBOL( constant_gaussian_kernel), p_gaussian_kernel, sizeof( float ) * 25 );

	delete[] p_gaussian_kernel;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Ŀ���� �����ϱ� �� �ʿ��� �ڷ�� �غ� �� Ŀ���� ������ ����̽��� ����
//	Shared_flag �Է� �� NO_SHARED �� SHARED �� �� ���� ��ũ�θ� ������
//	flag���� �´� Ŀ���� ����
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

float Do_Gaussian_on_GPU(IN unsigned char *p_bitmaps, OUT unsigned char *p_Gaussian, long width, long height, int Shared_flag)
{
	hipError_t cudaStatus;
	float device_time;
	CUDA_CALL(hipSetDevice(0));

	unsigned char* d_bitmaps;
	unsigned char*d_Gaussian;
	size_t mem_size;
	mem_size = width*height* sizeof(unsigned char);
	CUDA_CALL(hipMalloc(&d_bitmaps, mem_size));
	CUDA_CALL(hipMalloc(&d_Gaussian, mem_size));
	CUDA_CALL(hipMemcpy(d_bitmaps, p_bitmaps, mem_size, hipMemcpyHostToDevice));

	Set_Gaussain_Kernel();

	dim3 blockDIm(BLOCK_SIZE);
	dim3 gridDim(width*height/BLOCK_SIZE);
	CHECK_TIME_INIT_GPU();
	CHECK_TIME_START_GPU();

	switch (Shared_flag)
	{
	case NO_SHARED:
		Gaussian_kernel_no_shared << <gridDim, blockDIm >> > (d_bitmaps, d_Gaussian, width, height);
		break;
	case SHARED:
		Gaussian_kernel_shared << <gridDim, blockDIm, sizeof(unsigned char)* (BLOCK_SIZE + 2 *2)*5>> > (d_bitmaps, d_Gaussian, width, height);
		break;

	}
	CHECK_TIME_END_GPU(device_time);
	CUDA_CALL(cudaStatus = hipDeviceSynchronize());

	CHECK_TIME_DEST_GPU();
	CUDA_CALL(hipMemcpy(p_Gaussian, d_Gaussian, mem_size, hipMemcpyDeviceToHost));

	hipFree(d_bitmaps);
	hipFree(d_Gaussian);

	return device_time;
}