#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;
#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#define	IN
#define OUT
#define INOUT

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)


void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL(hipEventDestroy(cuda_timer_start));
	CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

#define N_SIZE (1 << 26)													// 전체 데이터 사이즈
#define NF_SIZE (1 << 6)													// Nf 크기

#define NO_SHARED 0															// shared memory를 사용하지 않는 커널 실행 flag
#define SHARED 1															// shared memory를 사용하는 커널 실행 flag

#define BLOCK_SIZE (1 << 6)													// CUDA 커널 thread block 사이즈

#define BLOCK_WIDTH (1 << 3)
#define BLOCK_HEIGHT (BLOCK_SIZE / BLOCK_WIDTH)

#define N_ITERATION (1 << 0)												// 실험 반복 횟수
extern __shared__ int shared_buffer[];
TIMER_T compute_time = 0;
TIMER_T device_time = 0;

int N;
int Nf;

int *h_ArrayElements;
int *h_SumOfArrayElements_CPU;
int *h_SumOfArrayElements_GPU_No_Shared;
int *h_SumOfArrayElements_GPU_Shared;

hipError_t Sum_n_elements_GPU(IN int *p_ArrayElements, OUT int *p_SumOfElements_GPU, int Nf, int Shared_flag);

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	배열의 index - Nf 부터 index + Nf 데이터 까지의 합을 계산하는 커널 코드
//	이 커널은 shared 메모리를 사용하지 않는다.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void Sum_n_elements_Kernel_No_shared(IN int *d_ArrayElements, OUT int *d_SumOfArrayElements, int N, int Nf) {
	const unsigned block_id = blockIdx.y * gridDim.x + blockIdx.x;
	const unsigned thread_id = threadIdx.y * blockDim.x + threadIdx.x;
	const unsigned id = block_id * BLOCK_SIZE + thread_id;

	for (int i = -Nf; i <= Nf; i++) {
		if (id + i >= N || id + i < 0) continue;
		d_SumOfArrayElements[id] += d_ArrayElements[id + i];
	}
	//만약 여기서 sum이라는 로컬변수를 두게되면 registor를 이용하는 것이 되기 때문에 속도가 빠름 따라서 정확히 측정하기위해선 위와 같이 해야함
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	배열의 index - Nf 부터 index + Nf 데이터 까지의 합을 계산하는 커널 코드
//	이 커널은 shared 메모리를 사용한다.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void Sum_n_elements_Kernel_shared(IN int *d_ArrayElements, OUT int *d_SumOfArrayElements, int N, int Nf) {
	const unsigned block_id	 = blockIdx.y * gridDim.x + blockIdx.x;
	const unsigned thread_id = threadIdx.y * blockDim.x + threadIdx.x;
	const unsigned id = block_id * BLOCK_SIZE + thread_id;
	int i;
	/*Todo*/

	if (thread_id == 0)
	{
		for (i = 0; i < Nf; i++)
		{
			if (id + i < Nf) shared_buffer[i] = 0;
			else shared_buffer[i] = d_ArrayElements[id + i - Nf];
		}
	}
	if (thread_id == BLOCK_SIZE-1)
	{
		for (i = 0; i <= Nf; i++)
		{
			if (id + i >= N) shared_buffer[thread_id+ i + Nf ] = 0;
			else shared_buffer[thread_id + i + Nf] = d_ArrayElements[id + i];
		}
	}
	shared_buffer[thread_id + Nf] = d_ArrayElements[id];
	__syncthreads();
	int sum = 0;
	for (i = 0; i <= 2*Nf; i++)
	{
		sum += shared_buffer[thread_id + i];
	}
	d_SumOfArrayElements[id] = sum;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	배열의 index - Nf 부터 index + Nf 데이터 까지의 합을 계산하는 C 코드
//	GPU kernel의 결과와 비교를 통해 옳은 계산을 하였는지 판단하는 데이터로 활용
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void Sum_n_elements_CPU(IN int *p_ArrayElements, OUT int *p_SumOfElements_CPU, int Nf) {
	int i, j, sum;

	for (i = 0; i < N; i++) {
		sum = 0;
		for (j = -Nf; j <= Nf; j++) {
			if (i + j >= N || i + j < 0) continue;
			sum += p_ArrayElements[i + j];
		}
		p_SumOfElements_CPU[i] = sum;
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	주어진 bin 파일을 읽는 코드
//	첫 4바이트는 전체 데이터의 개수, 다음 4바이트는 Nf의 크기, 그 이후 N개의 int형 데이터가 저장
//	데이터는 -100 ~ 100 까지의 범위 안의 정수
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void read_bin_file() {
	printf("***Binary File Read Start!!\n");
	FILE *fp = fopen("gen.bin", "rb");
	fread(&N, sizeof(int), 1, fp);
	fread(&Nf, sizeof(int), 1, fp);

	h_ArrayElements = (int *)malloc(N * sizeof(int));
	h_SumOfArrayElements_CPU = (int *)malloc(N * sizeof(int));
	h_SumOfArrayElements_GPU_No_Shared = (int *)malloc(N * sizeof(int));
	h_SumOfArrayElements_GPU_Shared = (int *)malloc(N * sizeof(int));

	fread(h_ArrayElements, sizeof(int), N, fp);

	fclose(fp);
	printf("***Binary File Read End!!\n\n");
}

void init_bin_file(IN int n, IN int nf) {
	printf("***Binary File Create Start!!\n");
	srand((unsigned)time(NULL));
	FILE *fp = fopen("gen.bin", "wb");
	fwrite(&n, sizeof(int), 1, fp);
	fwrite(&nf, sizeof(int), 1, fp);

	int i, input;

	for (i = 0; i < n; i++) {
		input = (int)((float)rand() / RAND_MAX * 200 - 100);
		fwrite(&input, sizeof(int), 1, fp);
	}

	fclose(fp);
	printf("***Binary File Create End!!\n\n");
}

int main()
{
	int i;
	init_bin_file(N_SIZE, NF_SIZE);
	read_bin_file();

	TIMER_T CPU_time = 0.0f, GPU_time_NO_SHARED = 0.0f, GPU_time_SHARED = 0.0f;

	for (i = 0; i < N_ITERATION; i++) {
		CHECK_TIME_START;
		Sum_n_elements_CPU(h_ArrayElements, h_SumOfArrayElements_CPU, Nf);
		CHECK_TIME_END(compute_time);
		CPU_time += compute_time;

		Sum_n_elements_GPU(h_ArrayElements, h_SumOfArrayElements_GPU_No_Shared, Nf, NO_SHARED);
		GPU_time_NO_SHARED += device_time;

		Sum_n_elements_GPU(h_ArrayElements, h_SumOfArrayElements_GPU_Shared, Nf, SHARED);
		GPU_time_SHARED += device_time;
	}

	for (i = 0; i < N; i++) {
		if (h_SumOfArrayElements_CPU[i] != h_SumOfArrayElements_GPU_No_Shared[i] || h_SumOfArrayElements_CPU[i] != h_SumOfArrayElements_GPU_Shared[i]) {
			printf("%d : CPU : %d,\tGPU no shared : %d,\tGPU shared : %d\n", i, h_SumOfArrayElements_CPU[i], h_SumOfArrayElements_GPU_No_Shared[i], h_SumOfArrayElements_GPU_Shared[i]);
			break;
		}
	}
	if (i == N)
		printf("***Kernel execution Success!!\n\n");

	printf("***CPU compute time : %.3f ms\n", CPU_time / N_ITERATION);
	printf("***GPU NO SHARED compute time : %.3f ms\n", GPU_time_NO_SHARED / N_ITERATION);
	printf("***GPU SHARED compute time : %.3f ms\n", GPU_time_SHARED / N_ITERATION);

	free(h_ArrayElements);
	free(h_SumOfArrayElements_CPU);
	free(h_SumOfArrayElements_GPU_No_Shared);
	free(h_SumOfArrayElements_GPU_Shared);

	return 0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	커널을 실행하기 전 필요한 자료들 준비 및 커널을 실행할 디바이스를 설정
//	Shared_flag 입력 시 NO_SHARED 나 SHARED 중 한 개의 매크로를 넣으면
//	flag값에 맞는 커널을 실행
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

hipError_t Sum_n_elements_GPU(IN int *p_ArrayElements, OUT int *p_SumOfElements_GPU, int Nf, int Shared_flag) {
	hipError_t cudaStatus;

	CUDA_CALL(hipSetDevice(0));

	int *d_ArrayElements, *d_SumOfElements;
	size_t mem_size;

	mem_size = N * sizeof(int);
	CUDA_CALL(hipMalloc(&d_ArrayElements, mem_size));
	CUDA_CALL(hipMalloc(&d_SumOfElements, mem_size));

	CUDA_CALL(hipMemcpy(d_ArrayElements, p_ArrayElements, mem_size, hipMemcpyHostToDevice));

	dim3 blockDIm(BLOCK_WIDTH, BLOCK_HEIGHT);
	dim3 gridDim(N / BLOCK_SIZE); 
	CHECK_TIME_INIT_GPU();
	CHECK_TIME_START_GPU();

	switch (Shared_flag)
	{
	case NO_SHARED:
		Sum_n_elements_Kernel_No_shared << <gridDim, blockDIm >> > (d_ArrayElements, d_SumOfElements, N, Nf);
		break;
	case SHARED:
		Sum_n_elements_Kernel_shared << <gridDim, blockDIm, sizeof(int)*(BLOCK_SIZE+2*Nf) >> > (d_ArrayElements, d_SumOfElements, N, Nf);
		break;
		
	}

	CUDA_CALL(cudaStatus = hipDeviceSynchronize());
	CHECK_TIME_END_GPU(device_time);
	CHECK_TIME_DEST_GPU();

	CUDA_CALL(hipMemcpy(p_SumOfElements_GPU, d_SumOfElements, mem_size, hipMemcpyDeviceToHost));
	
	hipFree(d_ArrayElements);
	hipFree(d_SumOfElements);

	return cudaStatus;
}
